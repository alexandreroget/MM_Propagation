#include "hip/hip_runtime.h"
#include "ComplexArraysContainerGPU.cuh"


__global__ void copyManyComplexArraysKernel(hipDoubleComplex* A, const unsigned int columnIndexA, const hipDoubleComplex* B, const unsigned int columnIndexB, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y;

  int indexA = (columnIndexA * (arraySize * numArrays)) + (idy * arraySize) + idx;
  int indexB = (columnIndexB * (arraySize * numArrays)) + (idy * arraySize) + idx;
  
  A[indexA] = B[indexB];
}


__global__ void copySingleComplexArrayKernel(hipDoubleComplex* A, const unsigned int rowIndexA, const unsigned int columnIndexA, const hipDoubleComplex* B, const unsigned int rowIndexB, const unsigned int columnIndexB, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  unsigned int indexA = (columnIndexA * (arraySize * numArrays)) + (rowIndexA * arraySize) + idx;
  unsigned int indexB = (columnIndexB * (arraySize * numArrays)) + (rowIndexB * arraySize) + idx;
  
  A[indexA] = B[indexB];
}


__global__ void addManyComplexArraysKernel(hipDoubleComplex* A, const unsigned int columnIndexA, const hipDoubleComplex* B, const unsigned int columnIndexB, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y;

  int indexA = (columnIndexA * (arraySize * numArrays)) + (idy * arraySize) + idx;
  int indexB = (columnIndexB * (arraySize * numArrays)) + (idy * arraySize) + idx;
  
  A[indexA] = hipCadd(A[indexA], B[indexB]);
}


__global__ void addSingleComplexArrayKernel(hipDoubleComplex* A, const unsigned int rowIndexA, const unsigned int columnIndexA, const hipDoubleComplex* B, const unsigned int rowIndexB, const unsigned int columnIndexB, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  unsigned int indexA = (columnIndexA * (arraySize * numArrays)) + (rowIndexA * arraySize) + idx;
  unsigned int indexB = (columnIndexB * (arraySize * numArrays)) + (rowIndexB * arraySize) + idx;
  
  A[indexA] = hipCadd(A[indexA], B[indexB]);
}


__global__ void multiplyManyComplexArraysKernel(hipDoubleComplex* A, const unsigned int columnIndexA, const hipDoubleComplex* B, const unsigned int columnIndexB, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y;
  
  int indexA = (columnIndexA * (arraySize * numArrays)) + (idy * arraySize) + idx;
  int indexB = (columnIndexB * (arraySize * numArrays)) + (idy * arraySize) + idx;
  
  A[indexA] = hipCmul(A[indexA], B[indexB]);
}


__global__ void multiplySingleComplexArrayKernel(hipDoubleComplex* A, const unsigned int rowIndexA, const unsigned int columnIndexA, const hipDoubleComplex* B, const unsigned int rowIndexB, const unsigned int columnIndexB, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  unsigned int indexA = (columnIndexA * (arraySize * numArrays)) + (rowIndexA * arraySize) + idx;
  unsigned int indexB = (columnIndexB * (arraySize * numArrays)) + (rowIndexB * arraySize) + idx;
  
  A[indexA] = hipCmul(A[indexA], B[indexB]);
}


__global__ void multiplyManyComplexArraysByScalarKernel(hipDoubleComplex* A, const unsigned int columnIndex, const hipDoubleComplex b, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y;

  int index = (columnIndex * (arraySize * numArrays)) + (idy * arraySize) + idx;
  
  A[index] = hipCmul(A[index], b);
}


__global__ void computeComplexConjugateKernel(hipDoubleComplex* A, const unsigned int columnIndexA, const hipDoubleComplex* B, const unsigned int columnIndexB, const unsigned int arraySize, const unsigned int numArrays) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y;

  int indexA = (columnIndexA * (arraySize * numArrays)) + (idy * arraySize) + idx;
  int indexB = (columnIndexB * (arraySize * numArrays)) + (idy * arraySize) + idx;
  
  A[indexA] = hipConj(B[indexB]);
}


ComplexArraysContainerGPU::ComplexArraysContainerGPU(unsigned int arraySize, unsigned int numRows, unsigned int numColumns) :
n(arraySize), nRows(numRows), nCols(numColumns) {
  unsigned int size = n * nRows * nCols;
  hipMalloc((void**)&data, sizeof(hipDoubleComplex) * size);
}


ComplexArraysContainerGPU::ComplexArraysContainerGPU(unsigned int arraySize, hipDoubleComplex value, unsigned int numRows, unsigned int numColumns) :
n(arraySize), nRows(numRows), nCols(numColumns) {
  unsigned int size = n * nRows * nCols;
  hipMalloc((void**)&data, sizeof(hipDoubleComplex) * size);
    
  hipDoubleComplex* host = new hipDoubleComplex[size];
  for(unsigned int i = 0 ; i < size ; i++) {
    host[i] = value;
  }
  
  hipMemcpy(data, host, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice);
  
  delete[] host;
}


ComplexArraysContainerGPU::~ComplexArraysContainerGPU() {
  hipFree(data);
}


void ComplexArraysContainerGPU::getDataFromCPU(ComplexArraysContainerCPU& source, const unsigned int columnIndex) {
  unsigned int size = n * nRows * nCols;
  hipDoubleComplex* host = new hipDoubleComplex[size];
  
  hipMemcpy(host, *this, sizeof(hipDoubleComplex) * size, hipMemcpyDeviceToHost);
  
  for(unsigned int p = 0 ; p < nRows ; p++) {
    unsigned int hostIndex = (columnIndex * nRows * nCols) + (p * n);
    for(unsigned int i = 0 ; i < n ; i++) {
      host[hostIndex] = make_hipDoubleComplex(source[p][i].real(), source[p][i].imag());
      hostIndex++;
    }
  }
  
  hipMemcpy(*this, host, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice);

  delete[] host;
}


void ComplexArraysContainerGPU::sendDataToCPU(ComplexArraysContainerCPU& target, const unsigned int columnIndex) const {
  unsigned int size = n * nRows * nCols;
  hipDoubleComplex* host = new hipDoubleComplex[size];
    
  hipMemcpy(host, *this, sizeof(hipDoubleComplex) * size, hipMemcpyDeviceToHost);
  
  for(unsigned int p = 0 ; p < nRows ; p++) {
    unsigned int hostIndex = (columnIndex * nRows * nCols) + (p * n);
    for(unsigned int i = 0 ; i < n ; i++) {
      target[p][i] = std::complex<double>(hipCreal(host[hostIndex]), hipCimag(host[hostIndex]));
      hostIndex++;
    }
  }
  
  delete[] host;
}


hipDoubleComplex* ComplexArraysContainerGPU::operator () () const { 
  return data; 
}


ComplexArraysContainerGPU::operator hipDoubleComplex* () const { 
  return data; 
}

