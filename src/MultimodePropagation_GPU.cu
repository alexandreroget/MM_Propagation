#include "hip/hip_runtime.h"
#include "MultimodePropagation_GPU.cuh"


MultimodePropagationGPU::MultimodePropagationGPU(const struct SimulationParameters& in) : MultimodePropagation(in) {
  setGPUThreadConfiguration();
}


MultimodePropagationGPU::~MultimodePropagationGPU() {
  hipfftDestroy(forward_plan_many);
  hipfftDestroy(inverse_plan_many);
}


void MultimodePropagationGPU::setGPUThreadConfiguration() {
  hipDeviceProp_t deviceProp;
  int deviceId = 0;
  hipGetDeviceProperties(&deviceProp, deviceId);

  n_threads = deviceProp.maxThreadsPerBlock;
  
  n_blocks = nt / n_threads;
  if ((nt % n_threads) != 0) n_blocks++;
}


void MultimodePropagationGPU::setFFTPlansMany() {
 int batch = M;
 int rank = 1;

 int nRows = nt;
 int n[1] = {nRows};

 int idist = nRows;
 int odist = nRows;

 int inembed[] = {nRows};
 int onembed[] = {nRows};

 int istride = 1;
 int ostride = 1;

 hipfftPlanMany(&forward_plan_many, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);
 hipfftPlanMany(&inverse_plan_many, rank, n, onembed, ostride, odist, inembed, istride, idist, HIPFFT_Z2Z, batch);
}


void MultimodePropagationGPU::computeLawsonRK(const unsigned int nz) {
  initLawsonRK();

  ComplexArraysContainerGPU psi_GPU(nt, M);
  psi_GPU.getDataFromCPU(psi);

  ComplexArraysContainerGPU E1_GPU(nt, M);
  E1_GPU.getDataFromCPU(E1);
  
  ComplexArraysContainerGPU E_GPU(nt, M, RK.s);
  for(unsigned int i = 0 ; i < RK.s ; i++) {
    E_GPU.getDataFromCPU(E[i], i);
  }
  for(unsigned int i = 0 ; i < nz ; i++) {
    // Compute $\psi_{n+1}^{p}$ 
    applyRungeKuttaMethod(psi_GPU, E_GPU, delta_z);
    
    computeExponentialOperatorsMatrix(E_GPU, E1_GPU);
    z += delta_z;
  }
  
  psi_GPU.sendDataToCPU(psi);
}


void MultimodePropagationGPU::computeExponentialOperatorsMatrix(ComplexArraysContainerGPU& E_GPU, const ComplexArraysContainerGPU& E1_GPU) {
  // Compute $E_{n+1,i}^{p} = E_{n,i}^{p} E_{1,i}^p$
  for(unsigned int i = 0 ; i < RK.s ; i++) {
    multiplyManyComplexArrays(E_GPU, E1_GPU, i, 0); 
  }
}


void MultimodePropagationGPU::applyRungeKuttaMethod(ComplexArraysContainerGPU& psi_GPU, const ComplexArraysContainerGPU& E_GPU, double h) {
  ComplexArraysContainerGPU N(nt, make_hipDoubleComplex(0., 0.), M, RK.s);

  ComplexArraysContainerGPU result(nt, M);
  // $\psi_{n+1}^{p} = psi^{p}$
  copyManyComplexArrays(result, psi_GPU);

  for(unsigned int i = 0 ; i < RK.s ; i++) {
    ComplexArraysContainerGPU psi_i(nt, M);
    copyManyComplexArrays(psi_i, psi_GPU);
    
    // $\psi_{n,i}^{p} = psi_n^{p} + h \sum_{j=0}^{i-1} (a_{i,j} N_{n,j})$
    if(i != 0.) {
      for(unsigned int j = 0 ; j < i ; j++) {
        if(RK.a[i][j] != 0.) {
          ComplexArraysContainerGPU sigma(nt, M);
          copyManyComplexArrays(sigma, N, 0, j);
          multiplyManyComplexArraysByScalar(sigma, make_hipDoubleComplex(RK.a[i][j]*h, 0.));
          addManyComplexArrays(psi_i, sigma);
        }
      }
    }
    
    // Compute $N(z_n + c_i h, \psi_{n,i})$
    computeNonlinearity(N, E_GPU, psi_i, i);
    
    // $\psi_{n+1}^{p} = \psi_{n+1}^{p} + b_i h N(z_n + c_i h, \psi_{n,i})$
    if(RK.b[i] != 0.) {
      ComplexArraysContainerGPU sigma(nt, M);
      copyManyComplexArrays(sigma, N, 0, i);
      multiplyManyComplexArraysByScalar(sigma, make_hipDoubleComplex(RK.b[i]*h, 0.));
      
      addManyComplexArrays(result, sigma);
    }
  }

  copyManyComplexArrays(psi_GPU, result);
}


void MultimodePropagationGPU::computeManyFFT(ComplexArraysContainerGPU& u) {
  hipfftExecZ2Z(forward_plan_many, u, u, HIPFFT_FORWARD);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::computeManyIFFT(ComplexArraysContainerGPU& u) {
  hipfftExecZ2Z(inverse_plan_many, u, u, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  multiplyManyComplexArraysByScalar(u, make_hipDoubleComplex(1./nt, 0.));
}


ComplexArraysContainerCPU MultimodePropagationGPU::getResult() {
  ComplexArraysContainerGPU Phi_GPU(nt, M);
  Phi_GPU.getDataFromCPU(psi);

  computeManyFFT(Phi_GPU);
  
  ComplexArraysContainerCPU Lawson_operator(nt, M);
  for(unsigned int p = 0 ; p < M ; p++) {
    Lawson_operator[p] = exp(L[p]*z);
  }
  
  ComplexArraysContainerGPU Lawson_operator_GPU(nt, M);
  Lawson_operator_GPU.getDataFromCPU(Lawson_operator);
  
  multiplyManyComplexArrays(Phi_GPU, Lawson_operator_GPU);
  
  computeManyIFFT(Phi_GPU);
  
  multiplyManyComplexArraysByScalar(Phi_GPU, make_hipDoubleComplex(1./conversion_factor, 0.));

  ComplexArraysContainerCPU Phi(nt, M);
  Phi_GPU.sendDataToCPU(Phi);

  return Phi;
}

// --------------------------------------------------------------------------------------------- //

MultimodePropagationGPU_RamanOFF::MultimodePropagationGPU_RamanOFF(const struct SimulationParameters& in) : MultimodePropagationGPU(in) {
  setFFTPlans();
}


void MultimodePropagationGPU_RamanOFF::setFFTPlans() {
  setFFTPlansMany();
}


void MultimodePropagationGPU_RamanOFF::computeNonlinearity(ComplexArraysContainerGPU& N, const ComplexArraysContainerGPU& E_GPU, const ComplexArraysContainerGPU& psi_i, const unsigned int i) {
  // Step 1: Compute $U^{p} = E_{n,i}^{p} \psi_{n,i}^{p}$
  ComplexArraysContainerGPU U(nt, M);
  copyManyComplexArrays(U, psi_i);
  computeManyFFT(U);
  multiplyManyComplexArrays(U, E_GPU, 0, i);
  computeManyIFFT(U);
  
  ComplexArraysContainerGPU U_conj(nt, M);
  computeComplexConjugate(U_conj, U);
  
  // Step 2: Compute $V^{l,m} = U^{l} \overline{U^{m}}$
  unsigned int n_pairs = unique_pairs.size();
  ComplexArraysContainerGPU V(nt, n_pairs);
  {
    unsigned int lm = 0;
    for(const auto& pair : unique_pairs) {
      unsigned int l = pair % M;
      unsigned int m = pair / M;
    
      copySingleComplexArray(V, U, lm, l);
      multiplySingleComplexArray(V, U_conj, lm, m);
          
      lm++;
    }
  }
    
  // Step 3: Compute $\sigma^{p} = \sum_{k,l,m} {\tilde{Q^{p}_{k,l,m}} (U^{k} V^{l,m}))}$
  ComplexArraysContainerGPU sigma(nt, make_hipDoubleComplex(0., 0.), M);
  
  for(unsigned int p = 0 ; p < M ; p++) {
    unsigned int num_nzv = Q_tilde[p].getNumberOfNonZeroValues();
    
    for(unsigned int j = 0 ; j < num_nzv ; j++) {
      NonZeroValue nzv = Q_tilde[p].getNonZeroValue(j);
      unsigned int k = nzv.index[0];
      unsigned int l = nzv.index[1];
      unsigned int m = nzv.index[2];
      double Q_pklm = nzv.value;
      
      auto it = std::find(unique_pairs.begin(), unique_pairs.end(), (l + m*M));
      unsigned int lm = std::distance(unique_pairs.begin(), it);
        
      // Compute $W^{k,l,m} = \tilde{Q^{p}_{k,l,m}} U^{k} V^{l,m}$
      ComplexArraysContainerGPU W(nt, 1);
      
      copySingleComplexArray(W, U, 0, k);
      multiplySingleComplexArray(W, V, 0, lm);
      multiplyManyComplexArraysByScalar(W, make_hipDoubleComplex(Q_pklm, 0.));
      
      addSingleComplexArray(sigma, W, p, 0);
    }
  }
  
  // Step 4: Compute $N_{n,i}^{p} = \ii\gamma \overline{E_{n,i}^{p}} \sigma^{p}$
  computeManyFFT(sigma);

  ComplexArraysContainerGPU E_i_conj(nt, M);
  computeComplexConjugate(E_i_conj, E_GPU, 0, i);
  multiplyManyComplexArrays(sigma, E_i_conj);
  
  computeManyIFFT(sigma);

  multiplyManyComplexArraysByScalar(sigma, make_hipDoubleComplex(sign_gamma, 0.));
  
  copyManyComplexArrays(N, sigma, i, 0);
}

// --------------------------------------------------------------------------------------------- //

MultimodePropagationGPU_RamanON::MultimodePropagationGPU_RamanON(const struct SimulationParameters& in) : MultimodePropagationGPU(in), fR(in.raman_proportion), hR_fft(nt, 1) {
  setFFTPlans();

  ComplexArraysContainerCPU hR_CPU(nt, 1);
  for(unsigned int i = 0 ; i < nt ; i++) {
    hR_CPU[0][i] = std::complex<double>(in.raman_response[i], 0.);
  }
  
  hR_fft.getDataFromCPU(hR_CPU);
  computeSingleFFT(hR_fft);
}


void MultimodePropagationGPU_RamanON::setFFTPlans() {
 setFFTPlansMany();
 hipfftPlan1d(&plan_1d, nt, HIPFFT_Z2Z, 1);
}


void MultimodePropagationGPU_RamanON::computeNonlinearity(ComplexArraysContainerGPU& N, const ComplexArraysContainerGPU& E_GPU, const ComplexArraysContainerGPU& psi_i, const unsigned int i) {
  // Step 1: Compute $U^{p} = E_{n,i}^{p} \psi_{n,i}^{p}$
  ComplexArraysContainerGPU U(nt, M);
  copyManyComplexArrays(U, psi_i);
  computeManyFFT(U);
  multiplyManyComplexArrays(U, E_GPU, 0, i);
  computeManyIFFT(U);
  
  ComplexArraysContainerGPU U_conj(nt, M);
  computeComplexConjugate(U_conj, U);
  
  // Step 2: Compute $V^{l,m} = U^{l} \overline{U^{m}}$ and $V_Raman^{l,m} = h_R^{T_0} \star (U^{l} \overline{U^{m}})$
  unsigned int n_pairs = unique_pairs.size();
  ComplexArraysContainerGPU V(nt, n_pairs);
  ComplexArraysContainerGPU V_Raman(nt, n_pairs);
  {
    unsigned int lm = 0;
    for(const auto& pair : unique_pairs) {
      unsigned int l = pair % M;
      unsigned int m = pair / M;
    
      copySingleComplexArray(V, U, lm, l);
      multiplySingleComplexArray(V, U_conj, lm, m);
    
      copySingleComplexArray(V_Raman, V, lm, lm);
      convolveWithRamanResponse(V_Raman, lm);
    
      lm++;
    }
  }  
  
  // Step 3: Compute $\sigma^{p} = \sum_{k,l,m} {\tilde{Q^{p}_{k,l,m}} U^{k} ((1 - f_R) V^{l,m} + (f_R T_0) V_Raman^{l,m})}$
  ComplexArraysContainerGPU sigma(nt, make_hipDoubleComplex(0., 0.), M);
  for(unsigned int p = 0 ; p < M ; p++) {
    unsigned int num_nzv = Q_tilde[p].getNumberOfNonZeroValues();
    
    for(unsigned int j = 0 ; j < num_nzv ; j++) {
      NonZeroValue nzv = Q_tilde[p].getNonZeroValue(j);
      unsigned int k = nzv.index[0];
      unsigned int l = nzv.index[1];
      unsigned int m = nzv.index[2];
      double Q_pklm = nzv.value;
      
      auto it = std::find(unique_pairs.begin(), unique_pairs.end(), (l + m*M));
      unsigned int lm = std::distance(unique_pairs.begin(), it);
      
      // Compute $W^{k,l,m} = \tilde{Q^{p}_{k,l,m}} U^{k} ((1 - f_R) V^{l,m} + (f_R T_0) V_Raman^{l,m})$
      ComplexArraysContainerGPU W(nt, 1);

      copySingleComplexArray(W, U, 0, k);
      multiplySingleComplexArray(W, V, 0, lm);
      multiplyManyComplexArraysByScalar(W, make_hipDoubleComplex(Q_pklm*(1-fR), 0.));

      addSingleComplexArray(sigma, W, p, 0);
      
      copySingleComplexArray(W, U, 0, k);
      multiplySingleComplexArray(W, V_Raman, 0, lm);
      multiplyManyComplexArraysByScalar(W, make_hipDoubleComplex(Q_pklm*(fR*T0), 0.));
      
      addSingleComplexArray(sigma, W, p, 0);
    }
  }
  
  // Step 4: Compute $N_{n,i}^{p} = \ii\gamma \overline{E_{n,i}^{p}} \sigma^{p}$
  computeManyFFT(sigma);

  ComplexArraysContainerGPU E_i_conj(nt, M);
  computeComplexConjugate(E_i_conj, E_GPU, 0, i);
  multiplyManyComplexArrays(sigma, E_i_conj);
  
  computeManyIFFT(sigma);

  multiplyManyComplexArraysByScalar(sigma, make_hipDoubleComplex(sign_gamma, 0.));
  
  copyManyComplexArrays(N, sigma, i, 0);
}


void MultimodePropagationGPU_RamanON::convolveWithRamanResponse(ComplexArraysContainerGPU& V_Raman, const unsigned int index) {
  ComplexArraysContainerGPU result(nt, 1);
  copySingleComplexArray(result, V_Raman, 0, index);

  computeSingleFFT(result);
  multiplyManyComplexArrays(result, hR_fft);
  computeSingleIFFT(result);
  
  multiplyManyComplexArraysByScalar(result, make_hipDoubleComplex(delta_t, 0.));
  
  copySingleComplexArray(V_Raman, result, index, 0);
}


void MultimodePropagationGPU_RamanON::computeSingleFFT(ComplexArraysContainerGPU& u) {
  hipfftExecZ2Z(plan_1d, u, u, HIPFFT_FORWARD);
}


void MultimodePropagationGPU_RamanON::computeSingleIFFT(ComplexArraysContainerGPU& u) {
  hipfftExecZ2Z(plan_1d, u, u, HIPFFT_BACKWARD);
  multiplyManyComplexArraysByScalar(u, make_hipDoubleComplex(1./nt, 0.));
}

// --------------------------------------------------------------------------------------------- //

void MultimodePropagationGPU::copyManyComplexArrays(ComplexArraysContainerGPU& arraysContainerA, const ComplexArraysContainerGPU& arraysContainerB, const unsigned int columnIndexA, const unsigned int columnIndexB) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, M, 1);

  copyManyComplexArraysKernel<<<grid, block>>>(arraysContainerA, columnIndexA, arraysContainerB, columnIndexB, nt, M);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::copySingleComplexArray(ComplexArraysContainerGPU& arraysContainerA, const ComplexArraysContainerGPU& arraysContainerB, const unsigned int rowIndexA, const unsigned int rowIndexB, const unsigned int columnIndexA, const unsigned int columnIndexB) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, 1, 1);

  copySingleComplexArrayKernel<<<grid, block>>>(arraysContainerA, rowIndexA, columnIndexA, arraysContainerB, rowIndexB, columnIndexB, nt, M);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::addManyComplexArrays(ComplexArraysContainerGPU& arraysContainerA, const ComplexArraysContainerGPU& arraysContainerB, const unsigned int columnIndexA, const unsigned int columnIndexB) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, M, 1);

  addManyComplexArraysKernel<<<grid, block>>>(arraysContainerA, columnIndexA, arraysContainerB, columnIndexB, nt, M);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::addSingleComplexArray(ComplexArraysContainerGPU& arraysContainerA, const ComplexArraysContainerGPU& arraysContainerB, const unsigned int rowIndexA, const unsigned int rowIndexB, const unsigned int columnIndexA, const unsigned int columnIndexB) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, 1, 1);

  addSingleComplexArrayKernel<<<grid, block>>>(arraysContainerA, rowIndexA, columnIndexA, arraysContainerB, rowIndexB, columnIndexB, nt, M);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::multiplyManyComplexArrays(ComplexArraysContainerGPU& arraysContainerA, const ComplexArraysContainerGPU& arraysContainerB, const unsigned int columnIndexA, const unsigned int columnIndexB) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, M, 1);

  multiplyManyComplexArraysKernel<<<grid, block>>>(arraysContainerA, columnIndexA, arraysContainerB, columnIndexB, nt, M);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::multiplySingleComplexArray(ComplexArraysContainerGPU& arraysContainerA, const ComplexArraysContainerGPU& arraysContainerB, const unsigned int rowIndexA, const unsigned int rowIndexB, const unsigned int columnIndexA, const unsigned int columnIndexB) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, 1, 1);

  multiplySingleComplexArrayKernel<<<grid, block>>>(arraysContainerA, rowIndexA, columnIndexA, arraysContainerB, rowIndexB, columnIndexB, nt, M);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::multiplyManyComplexArraysByScalar(ComplexArraysContainerGPU& arraysContainerA, const hipDoubleComplex complexScalar, const unsigned int columnIndexA) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, M, 1);
  
  multiplyManyComplexArraysByScalarKernel<<<grid, block>>>(arraysContainerA, columnIndexA, complexScalar, nt, M);
  hipDeviceSynchronize();
}


void MultimodePropagationGPU::computeComplexConjugate(ComplexArraysContainerGPU& arraysContainerA, const ComplexArraysContainerGPU& arraysContainerB, const unsigned int columnIndexA, const unsigned int columnIndexB) {
  dim3 block(n_threads, 1, 1);
  dim3 grid(n_blocks, M, 1);
  
  computeComplexConjugateKernel<<<grid, block>>>(arraysContainerA, columnIndexA, arraysContainerB, columnIndexB, nt, M);
  hipDeviceSynchronize();
}

