#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include "MultimodePropagation_GPU.cuh"

ComplexArray build_soliton(const double t_final, const unsigned int nt, const double t0, const double c, const double q);

// ---------------------------------------------------------------------------- //

int main() {
  unsigned int nt = (int) pow(2,15);
  double t_final = 160.; // in ps

  double t0 = -t_final/10;
  double c = 1.;

  // ----------- Set up input parameters ----------- //
  
  struct SimulationParameters in;
  
  in.n_modes = 2;
  
  in.dispersion_coefficients = std::vector<std::vector<double>>(2, std::vector<double>(3, 0.));
  in.dispersion_coefficients[0][2] = 18.938e-3;
  in.dispersion_coefficients[1][2] = 18.866e-3;
  
  double SR = 6.4341e12;
  
  Sparse3DArray Q0;
  struct NonZeroValue nzv;
  
  nzv = {0, 0, 0, SR};
  Q0.addNonZeroValue(nzv);
  nzv = {0, 0, 1, SR/3};
  Q0.addNonZeroValue(nzv);
  nzv = {0 , 1, 1, SR/2};
  Q0.addNonZeroValue(nzv);
  // nzv = {1, 1, 1, -SR/3};
  // Q0.addNonZeroValue(nzv);
  in.coupling_coefficients.push_back(Q0);
  
  Sparse3DArray Q1;
  // nzv = {0, 0, 0, -SR/3};
  // Q1.addNonZeroValue(nzv);
  nzv = {1, 0, 0, SR/5};
  Q1.addNonZeroValue(nzv);
  nzv = {1, 1, 0, SR/3};
  Q1.addNonZeroValue(nzv);
  nzv = {1, 1, 1, 0.8*SR};
  Q1.addNonZeroValue(nzv);
  in.coupling_coefficients.push_back(Q1);
  
  in.fiber_length = 16.;

  in.n_steps = 1024;
  
  in.method_order = 4;

  double lambda = 1030e-9; // central wavelength in m
  double f0 = 2.99792458e-4/lambda;
  double speed_of_light = 2.99792458e-4; // speed of light in m/ps
  double w0 = 2*M_PI*f0;
  double n2 = 2.3*1e-20; // m^2 W^-1
  in.nonlinearity_const = (n2*w0)/speed_of_light;
  
  in.raman_proportion = 0.18;
  
  in.raman_response = std::vector<double>(nt);
  double tau[2] = {12.2e-3, 32e-3};
  double delta = t_final/nt;
  double a = (1./(tau[0]*tau[0]) + 1./(tau[1]*tau[1])) * tau[0];
  for(unsigned int i = 0 ; i < nt ; i++) {
    double t = i * delta;
    double b = exp(-t/tau[1]) * sin(t/tau[0]);
    in.raman_response[i] = a*b;
  }

  double beta_2 = in.dispersion_coefficients[0][2];

  in.nt = nt;
  in.pulse_width = std::sqrt(beta_2)/(SR*in.nonlinearity_const);
  in.time_window = t_final;

  in.initial_fields = ComplexArraysContainer(nt, 2);
  in.initial_fields[0] = build_soliton(t_final/std::sqrt(beta_2/2), nt, t0/std::sqrt(beta_2/2), c*std::sqrt(2/beta_2), SR*in.nonlinearity_const);
  in.initial_fields[1] = build_soliton(t_final/std::sqrt(beta_2/2), nt, -t0/std::sqrt(beta_2/2), -c*std::sqrt(2/beta_2), 0.8*SR*in.nonlinearity_const);
  
  std::ofstream outputFile("GRIN2_GPU_RamanON.txt");
  
  if (!outputFile.is_open()) {
    std::cerr << "Erreur : Impossible d'ouvrir le fichier de sortie." << std::endl;
    return 1;
  }

  MultimodePropagation* mm_propagation = new MultimodePropagationGPU_RamanON(in);
  
  mm_propagation->computeLawsonRK(in.n_steps);
  
  ComplexArraysContainer Phi_out(nt, 2);
  Phi_out = mm_propagation->getResult();
  
  for(unsigned int i = 0 ; i < nt ; i++) {
    outputFile << std::abs(in.initial_fields[0][i]) << " ; " << std::abs(in.initial_fields[1][i]) << " ; " << std::abs(Phi_out[0][i]) << " ; " << std::abs(Phi_out[1][i]) << std::endl;
  }
  
  outputFile.close();
  
  delete mm_propagation;

  return 0;
}

// ---------------------------------------------------------------------------- //

ComplexArray build_soliton(const double t_final, const unsigned int nt, const double t0, const double c, const double q) {
  ComplexArray phi(nt);
  
  std::complex<double> i(0.,1.);

  double h = (double) t_final/nt;
  double t = -t_final/2;

  for(unsigned int j = 0 ; j < nt ; j++) {
    phi[j] = 0.5 * std::sqrt(q/2) * (1./std::cosh(q/4 * (t-t0))) *  std::exp(i*c*((t-t0)/2));
    t += h;
  }
  return phi;
}

